//-------------------------------------------------------------------------
// canonical error handling (implementation)
//-------------------------------------------------------------------------
#include "gpu_bf_error_handling.cuh"

//-------------------------------------------------------------------------
// CUDA runtime API
//-------------------------------------------------------------------------
void _checkCudaRTErrors( const hipError_t result, const char* const str_command, const char* const str_filename, int const index_line )
{
	if( hipSuccess != result )
	{

		// print error location and CUDA error strings
		mexPrintf( "\nCUDA error in line %d of file \"%s\":\n", index_line, str_filename );
		mexPrintf( "\t\"%s\"\n", str_command );
		mexPrintf( "\tcode = %d (%s): %s\n", static_cast<unsigned int>( result ), hipGetErrorName( result ), hipGetErrorString( result ) );

		// reset device to clean memory before exit
		hipDeviceReset();

		// print error message, exit program
		mexErrMsgIdAndTxt( "FMM:ErrorCUDA", "CUDA error!" );

	}
} // void _checkCudaRTErrors( const hipError_t result, const char* const str_command, const char* const str_filename, int const index_line )

//-------------------------------------------------------------------------
// cuFFT library
//-------------------------------------------------------------------------
void _checkCudaFFTErrors( const hipfftResult_t result, const char* const str_command, const char* const str_filename, int const index_line )
{
	if( HIPFFT_SUCCESS != result )
	{

		// print error location and CUDA error strings
		mexPrintf( "CUDA error at %s:%d:\n", str_filename, index_line );
		mexPrintf( "\t\"%s\"\n", str_command );
		mexPrintf( "code = %d\n", static_cast<unsigned int>( result ) );

		// reset device to clean memory before exit
		hipDeviceReset();

		// print error message, exit program
		mexErrMsgIdAndTxt( "FMM:ErrorCUDA", "CUDA error!" );

	}
} // void _checkCudaFFTErrors( const hipError_t result, const char* const str_command, const char* const str_filename, int const index_line )
