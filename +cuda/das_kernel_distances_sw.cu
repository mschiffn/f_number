#include "hip/hip_runtime.h"
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
// DAS kernel, propagation distances of spherical wave w/ virtual source
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
// author: Martin F. Schiffner
// date: 2025-08-23
// modified: 2025-08-23

__global__ void das_kernel_distances_sw( t_float_gpu* const distances,
                                         const t_float_gpu* const positions_x, const t_float_gpu* const positions_z, const int N_positions_x, const int N_positions_z,
                                         const t_float_gpu pos_tx_x, const t_float_gpu pos_tx_z )
{

	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// 0.) local variables
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// a) thread indices
	const int l_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int l_z = blockIdx.y * blockDim.y + threadIdx.y;

	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// 1.) compute propagation distances
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// ensure validity of positions
	if( l_x < N_positions_x && l_z < N_positions_z )
	{
		float distance_x = positions_x[ l_x ] - pos_tx_x;
		float distance_z = positions_z[ l_z ] - pos_tx_z;
		distances[ l_x * N_positions_z + l_z ] = __fsqrt_rn( distance_x * distance_x + distance_z * distance_z );
	}

}