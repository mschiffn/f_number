#include "hip/hip_runtime.h"
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
// DAS kernel, propagation distances of steered PW
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
// author: Martin F. Schiffner
// date: 2023-12-21
// modified: 2023-12-21

__global__ void das_kernel_distances_pw( t_float_gpu* const distances,
                                         const t_float_gpu* const positions_x, const t_float_gpu* const positions_z, const int N_positions_x, const int N_positions_z,
                                         const t_float_gpu e_steering_x, const t_float_gpu e_steering_z, const t_float_gpu pos_tx_ctr_x_ref )
{

	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// 0.) local variables
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// a) thread indices
	const int l_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int l_z = blockIdx.y * blockDim.y + threadIdx.y;

	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// 1.) compute propagation distances
	//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// ensure validity of positions
	if( l_x < N_positions_x && l_z < N_positions_z )
	{
		// b) argument for complex exponential in inverse DFT
		distances[ l_x * N_positions_z + l_z ] = e_steering_x * ( positions_x[ l_x ] - pos_tx_ctr_x_ref ) + e_steering_z * positions_z[ l_z ];
	}

}